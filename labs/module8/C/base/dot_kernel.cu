#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void dot_kernel(int *A, int *B, int *C, int m, int n)
{

	extern __shared__ int temp[];
	int i = blockIdx.x;
	int j = threadIdx.x;

	if ( (i < m) && (j < n) ) temp[j] = A[i*n+j] * B[i*n+j];
	__syncthreads();

	int k = j + blockDim.x;
	while ( (i < m) && (k < n) ) 
	{
		temp[j] += A[i*n+k] * B[i*n+k];
		k += blockDim.x;
	}
	__syncthreads();

	for (int stride = blockDim.x/2; stride >  0; stride /= 2) {
		if (j < stride)
			temp[j] += temp[j + stride];
		__syncthreads();
	}

	C[blockIdx.x] = temp[0];
}


extern "C" void dot_cuda(int *A, int *B, int *C, int m, int n)
{
	int *A_d, *B_d, *C_d, size;
	hipMalloc((void **)&A_d, m*n*sizeof(int));
	hipMalloc((void **)&B_d, m*n*sizeof(int));
	hipMalloc((void **)&C_d, m*sizeof(int));

	hipMemcpy(A_d, A, m*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, m*n*sizeof(int), hipMemcpyHostToDevice);

	size = min(512,n);

	dot_kernel<<<m,size,size*sizeof(int)>>>(A_d,B_d,C_d,m,n);
	hipDeviceSynchronize();

	hipMemcpy(C, C_d, m*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(A_d); hipFree(B_d); hipFree(C_d);

}
