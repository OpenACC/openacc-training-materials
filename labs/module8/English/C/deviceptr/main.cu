#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" void dot_acc(int*, int*, int*, int, int);
extern "C" void dot(int*, int*, int*, int, int);

int main()
{

	int i, j, m, n;
	int *A, *B, *C, *D;
	int *A_d, *B_d, *C_d;

	srand(0);

	m = 4098;
	n = 4098;

	A = (int*) malloc( m*n * sizeof(int));
	B = (int*) malloc( m*n * sizeof(int));
	C = (int*) malloc(  m  * sizeof(int));
	D = (int*) malloc(  m  * sizeof(int));

	for( i = 0; i < m; i++ ) {
		for( j = 0; j < n; j++ ) {
			A[i*n+j] = rand() % 100 + 1;
			B[i*n+j] = rand() % 100 + 1;
		}
	}

	hipMalloc((void **)&A_d, m*n*sizeof(int));
	hipMalloc((void **)&B_d, m*n*sizeof(int));
	hipMalloc((void **)&C_d, m*  sizeof(int));

	hipMemcpy(A_d, A, m*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, m*n*sizeof(int), hipMemcpyHostToDevice);

	dot_acc(A_d,B_d,C_d,m,n);

	hipMemcpy(C, C_d, m*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(A_d); hipFree(B_d); hipFree(C_d);

	dot(A,B,D,m,n);

	for( i = 0; i < m; i++ ) {
		if( C[i] != D[i] ) {
			printf("Error at index %i\n", i);
			return 0;
		}
	}

	printf("Program finished sucessfully.\n");
	return 0;

}
