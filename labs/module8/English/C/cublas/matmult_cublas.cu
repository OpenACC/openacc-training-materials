#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" void matmult_cublas(float*, float*, float*, int, int, int);

void matmult_cublas(float *A, float *B, float *C, int m, int k, int n)
{
	int lda=m, ldb=k, ldc=m;
	const float alf = 1.0;
	const float bet = 0.0;

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
		m, n, k, &alf, A, lda, B, ldb, &bet, C, ldc);

	hipblasDestroy(handle);

	return;

}
